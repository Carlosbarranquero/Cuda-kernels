
#include <hip/hip_runtime.h>
__global__
void dot_kernel(float* A, float* B, float* C, const int B_rows_number, const int C_rows_number)
{
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    int col = blockIdx.y * blockDim.y + threadIdx.y;

    int thread_id = row*C_rows_number + col; // col major

    float tmpSum = 0;

    if(thread_id < B_rows_number*C_rows_number)
    {
        for(int i=0; i<B_rows_number; i++)
        {
            tmpSum += A[thread_id%C_rows_number + i*C_rows_number] * B[thread_id/C_rows_number*B_rows_number + i];
        }

        C[thread_id] = tmpSum;    
    }
}


__global__
void convolve_kernel(float* input_dev,float* kernel_dev, float* result_dev,const int result_rows_number, const int result_cols_number, const int input_dim, const int kernel_dim)
{
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    int col = blockIdx.y * blockDim.y + threadIdx.y;

    int thread_id = row*result_rows_number + col; // col major

    if(thread_id < result_cols_number*result_rows_number)
    {
        int offset = (input_dim - result_rows_number) * (thread_id/result_rows_number);
    
        float tmp_result = 0.0;

        for(int i=0; i<kernel_dim; i++)
        {
            for(int j=0; j<kernel_dim; j++)
            {
                tmp_result += input_dev[thread_id + offset + i*input_dim + j] * kernel_dev[i*kernel_dim + j];
            }
        }
        result_dev[thread_id] = tmp_result;

    }

}


__global__
void convolve_kernel_3d(const float* input_dev, const float* kernel_dev, float* result_dev, const int result_rows_number, const int result_cols_number, const int input_dim, const int kernel_dim, const int channels)
{
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    int col = blockIdx.y * blockDim.y + threadIdx.y;

    int thread_id = row*result_rows_number + col; // col major

    int input_size = (result_rows_number +  kernel_dim - 1)*(result_rows_number +  kernel_dim - 1);

    if(thread_id < result_cols_number*result_rows_number)
    {
        int offset = (input_dim - result_rows_number) * (thread_id/result_rows_number);

        float tmp_result = 0.0;

        for(int k=0; k < channels; k++) // this can be optimazed with share memory 
        {
            for(int i=0; i<kernel_dim; i++)
            {
                for(int j=0; j<kernel_dim; j++)
                {
                    tmp_result += input_dev[thread_id + offset + i*input_dim + j + input_size*k] * kernel_dev[i*kernel_dim + j + kernel_dim*kernel_dim*k];
                }
            }
        }
        result_dev[thread_id] = tmp_result;
    }

}

__global__
void softmax_kernel(int n, int rows_number, float* inputs, float* outputs)
{
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    int col = blockIdx.y * blockDim.y + threadIdx.y;

    int thread_id = row*rows_number + col; // col major

    const int columns_number = n/rows_number;

    if(thread_id < rows_number*columns_number) 
    {
        float sum = 0;

        for(int j = 0; j < columns_number; j++)
        {
            sum += inputs[thread_id%rows_number + rows_number*j];
        }

        outputs[thread_id] = inputs[thread_id]/sum;
    }

}
