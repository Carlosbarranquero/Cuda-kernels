#include <stdlib.h>
#include <iostream>
#include "cuda_operations.cu"
using namespace std;

void print_host(float* vector, const int& size, const int& rows)
{
    const int cols = size/rows; // col_major

    for (int i=0; i<rows; i++)
    {
        for (int j=0; j<cols; j++)
        {
            cout<<*(vector + i + rows*j )<< " ";
        }

        cout<<" "<<endl;
    }
}

int main()
{
    
    //  Test convolve 
    cout<<"<-----------------Test convolve----------------->"<<endl;

    const int input_rows_number = 5;
    const int input_cols_number = 5;

    const int kernel_rows_number = 3; 
    const int kernel_cols_number = 3;

    const int result_rows_number = (input_rows_number - kernel_rows_number + 1);
    const int result_cols_number = (input_cols_number - kernel_cols_number + 1);

    const size_t input_size = input_cols_number*input_rows_number;
    const size_t kernel_size = kernel_cols_number*kernel_rows_number;
    const size_t result_size = result_rows_number*result_cols_number;

    //host 

    float* input_host = nullptr;
    float* kernel_host = nullptr;
    float* result_host = nullptr;

    //Malloc

    if(hipHostMalloc(&input_host, input_size*sizeof(float)) != hipSuccess)
        cout << "input_host allocation error" << endl;
    if(hipHostMalloc(&kernel_host, kernel_size*sizeof(float)) != hipSuccess)
        cout << "kernel_host allocation error" << endl;
    if(hipHostMalloc(&result_host, result_size*sizeof(float)) != hipSuccess)
        cout << "result_host allocation error" << endl;

    //Set values

    for (int i=0; i< input_size; i++)
        *(input_host + i) = i;

    for (int i=0; i< kernel_size; i++)
        *(kernel_host + i) = 2.0;

    for (int i=0; i< result_size; i++)
        *(result_host + i) = 0.0;

    //Print

    cout<<"input host"<<endl;
    print_host(input_host, input_size, input_rows_number);
    cout<<""<<endl;
    cout<<"kernel host "<<endl;
    print_host(kernel_host, kernel_size, kernel_rows_number);
    cout<<""<<endl;
    
    //dev

    float* input_dev = nullptr;
    float* kernel_dev = nullptr;
    float* result_dev = nullptr;

    //Malloc

    if(hipMalloc(&input_dev, input_size*sizeof(float)) != hipSuccess)
        cout << "Cuda malloc error" << endl;
    if(hipMalloc(&kernel_dev, kernel_size*sizeof(float)) != hipSuccess)
        cout << "Cuda malloc error" << endl;
    if(hipMalloc(&result_dev, result_size*sizeof(float)) != hipSuccess)
        cout << "Cuda malloc error" << endl;

    //Cpy values H -> D

    if(hipMemcpy(input_dev, input_host, input_size*sizeof(float), hipMemcpyHostToDevice) != hipSuccess)
        cout << "input_dev error" << endl;
    if(hipMemcpy(kernel_dev, kernel_host, kernel_size*sizeof(float), hipMemcpyHostToDevice) != hipSuccess)
        cout << "kernel_dev error" << endl;
    if(hipMemcpy(result_dev, result_host, result_size*sizeof(float), hipMemcpyHostToDevice) != hipSuccess)
        cout << "result_dev error" << endl;

    int threads = 32; // 64
    int blocks = (result_size + threads - 1 ) / threads;

    cout<<"threadsPerBlock: "<<threads<<endl;
    cout<<"blocksPerGrid: "<<blocks<<endl;

    dim3 threadsPerBlock(threads, threads);
    dim3 blocksPerGrid(blocks, blocks);

    convolve_kernel<<<blocksPerGrid, threadsPerBlock>>>(input_dev, kernel_dev, result_dev, result_rows_number, result_cols_number, input_rows_number, kernel_rows_number);
    
    if(hipMemcpy(result_host, result_dev, result_size*sizeof(float), hipMemcpyDeviceToHost) != hipSuccess)
        cout << "Cuda matrix memcpy error" << endl;

    cout<<""<<endl;
        
    print_host(result_host, result_size, result_rows_number);

//  Test dot A*B  = C

    cout<<""<<endl;
    cout<<"<-----------------Test dot----------------->"<<endl;

    const int A_rows_number = 10;
    const int A_cols_number = 15;
    const int A_size = A_rows_number*A_cols_number;

    const int B_rows_number = 15;
    const int B_cols_number = 20;
    const int B_size = B_rows_number*B_cols_number;

    const int C_rows_number = A_rows_number;
    const int C_cols_number = B_cols_number;
    const int C_size = C_rows_number*C_cols_number;

    //host 

    float* A_host = nullptr;
    float* B_host = nullptr;
    float* C_host = nullptr;

    //Malloc

    if(hipHostMalloc(&A_host, A_size*sizeof(float)) != hipSuccess)
        cout << "A_host allocation error" << endl;
    if(hipHostMalloc(&B_host, B_size*sizeof(float)) != hipSuccess)
        cout << "B_host allocation error" << endl;
    if(hipHostMalloc(&C_host, C_size*sizeof(float)) != hipSuccess)
        cout << "C_host allocation error" << endl;

    //Set values

    for (int i=0; i< A_size; i++)
        *(A_host + i) = rand() % 10;

    for (int i=0; i< B_size; i++)
        *(B_host + i) = rand() % 10;

    for (int i=0; i< C_size; i++)
        *(C_host + i) = 0.0;
    
    //Print

    cout<<"A host"<<endl;
    print_host(A_host, A_size, A_rows_number);
    cout<<""<<endl;
    cout<<"B host "<<endl;
    print_host(B_host, B_size, B_rows_number);
    cout<<""<<endl;
    
    //dev

    float* A_dev = nullptr;
    float* B_dev = nullptr;
    float* C_dev = nullptr;

    //Malloc

    if(hipMalloc(&A_dev, A_size*sizeof(float)) != hipSuccess)
        cout << "Cuda malloc error" << endl;
    if(hipMalloc(&B_dev, B_size*sizeof(float)) != hipSuccess)
        cout << "Cuda malloc error" << endl;
    if(hipMalloc(&C_dev, C_size*sizeof(float)) != hipSuccess)
        cout << "Cuda malloc error" << endl;

    //Cpy values H -> D

    if(hipMemcpy(A_dev, A_host, A_size*sizeof(float), hipMemcpyHostToDevice) != hipSuccess)
        cout << "A_dev copy error" << endl;
    if(hipMemcpy(B_dev, B_host, B_size*sizeof(float), hipMemcpyHostToDevice) != hipSuccess)
        cout << "B_dev copy error" << endl;
    if(hipMemcpy(C_dev, C_host, C_size*sizeof(float), hipMemcpyHostToDevice) != hipSuccess)
        cout << "C_dev copy error" << endl;
    
    int threads_ = 32; // 64
    int blocks_ = (C_size + threads - 1 ) / threads;

    cout<<"threadsPerBlock: "<<threads_<<endl;
    cout<<"blocksPerGrid: "<<blocks_<<endl;

    dim3 threadsPerBlock_dot(threads_, threads_);
    dim3 blocksPerGrid_dot(blocks_, blocks_);

    dot_kernel<<<blocksPerGrid_dot, threadsPerBlock_dot>>>(A_dev, B_dev, C_dev, B_rows_number, C_rows_number);

    if(hipMemcpy(C_host, C_dev, C_size*sizeof(float), hipMemcpyDeviceToHost) != hipSuccess)
        cout << "C memcpy error" << endl;

    cout<<""<<endl;
        
    print_host(C_host, C_size, C_rows_number);

}
